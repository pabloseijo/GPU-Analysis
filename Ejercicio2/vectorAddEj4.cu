/**
 * Suma dos vectores: C = A + B (Medición de tiempos por etapas)
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define checkError(ans) { asserError((ans), __FILE__, __LINE__); }
inline void asserError(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"Error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define TSET(time)  clock_gettime( CLOCK_MONOTONIC, &(time) )
#define TINT(ts,te) { ( (double) 1000.*( (te).tv_sec - (ts).tv_sec ) + ( (te).tv_nsec - (ts).tv_nsec )/(double) 1.e6 ) }

#define MAX_TH_PER_BLOCK 1024
#define NELDEF 1000
#define TPBDEF 256
#define NREPDEF 1

typedef float basetype;

/**
 * Código CUDA
 */
__global__ void vectorAdd(const basetype *A, const basetype *B, basetype *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(int argc, char *argv[]) {
    basetype *h_A = NULL, *h_B = NULL, *h_C = NULL, *h_C2 = NULL;
    basetype *d_A = NULL, *d_B = NULL, *d_C = NULL;
    unsigned int numElements = (argc > 1) ? atoi(argv[1]) : NELDEF;
    unsigned int tpb = (argc > 2) ? atoi(argv[2]) : TPBDEF;
    unsigned int nreps = (argc > 3) ? atoi(argv[3]) : NREPDEF;
    size_t size = numElements * sizeof(basetype);
    struct timespec tstart, tend;
    double tint;

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 threadsPerBlock(tpb);
    dim3 blocksPerGrid((numElements + threadsPerBlock.x - 1) / threadsPerBlock.x);

    printf("Suma de vectores de %u elementos (%u reps), con %u bloques de %u threads\n",
           numElements, nreps, blocksPerGrid.x, threadsPerBlock.x);

    // --- Reserva de memoria en el host ---
    h_A = (basetype *)malloc(size);
    h_B = (basetype *)malloc(size);
    h_C = (basetype *)malloc(size);
    h_C2 = (basetype *)malloc(size);

    // Inicialización de vectores
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (basetype)RAND_MAX;
        h_B[i] = rand() / (basetype)RAND_MAX;
    }

    // --- (a) Reserva de memoria en la GPU ---
    hipEventRecord(start, 0);
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU: Tiempo de reserva de memoria: %.2f ms\n", elapsedTime);

    // --- (b) Copia de vectores Host -> Device ---
    hipEventRecord(start, 0);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU: Tiempo de copia Host -> Device: %.2f ms\n", elapsedTime);

    // --- (c) Ejecución del kernel ---
    hipEventRecord(start, 0);
    for (unsigned int r = 0; r < nreps; ++r) {
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU: Tiempo de ejecución del kernel: %.2f ms\n", elapsedTime);

    // --- (d) Copia de resultados Device -> Host ---
    hipEventRecord(start, 0);
    hipMemcpy(h_C2, d_C, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU: Tiempo de copia Device -> Host: %.2f ms\n", elapsedTime);

    // Verificación de resultados
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_C2[i] - (h_A[i] + h_B[i])) > 1e-5) {
            fprintf(stderr, "Error en la verificación del elemento %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Suma correcta.\n");

    // Liberar memoria
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C2);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Terminamos\n");
    return 0;
}
