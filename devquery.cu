
#include <hip/hip_runtime.h>
#include <stdio.h>

void printDevProp(hipDeviceProp_t devProp)
{
// TODO: Completar esta función
}

int main(int argc, char *argv[]) {
  int numDevs;
  hipDeviceProp_t prop;
  hipError_t error;

  // Obtiene el número de dispositivos (tarjetas GPUs disponibles)
  error = hipGetDeviceCount(&numDevs);
  if(error != hipSuccess) {
    fprintf(stderr, "Error obteniendo numero de dispositivos: %s en %s linea %d\n", hipGetErrorString(error), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }

  printf("Numero de dispositivos = %d\n", numDevs);

  // Recorre las tarjetas disponibles y obtiene las propiedades de las mismas en prop.
  for(int i=0; i < numDevs; i++) {
    error = hipGetDeviceProperties(&prop, i);
    if(error != hipSuccess) {
      fprintf(stderr, "Error obteniendo propiedades del dispositivo %d: %s en %s linea %d\n", i, hipGetErrorString(error), __FILE__, __LINE__);
      exit(EXIT_FAILURE);
    }
    printf("\nDispositivo #%d\n", i);
    printDevProp(prop);
  }

  return(EXIT_SUCCESS);
}
